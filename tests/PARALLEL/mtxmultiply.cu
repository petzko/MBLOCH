
#include <hip/hip_runtime.h>
__global__ void mtx_multiply(const double2 * A, const double2 * B, double2 * C,
	const int N, const int K, const int L ){


	int col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;

	if (col >= L || row >= K) 
		return;

	C[row*L + col].x = 0.0;
	C[row*L + col].y = 0.0;
	for(int k = 0; k < N ; k++){

		C[row*L + col].x += A[row*N+k].x*B[k*L+col].x - A[row*N+k].y*B[k*L+col].y;
		C[row*L + col].y += A[row*N+k].x*B[k*L+col].y + A[row*N+k].y*B[k*L+col].x;

	}
}