#include <hip/hip_runtime.h>

__global__ void make_step(const double* D_N, const double2 * V, const double2* P,
							const double* c,
							const double dt, const int N, const int m, const int store_idx, 
							const int * order, 
							double2* F , double2* V_new)

{

	int t = threadIdx.x + blockDim.x*blockIdx.x;

	if(t >= N || t < 0)
		return;

	F[t*m + store_idx] = P[t];

	for(int k = 0; k < N; k++){
		F[t*m + store_idx].x += D_N[t*N+k]*V[k].x;
     	F[t*m + store_idx].y += D_N[t*N+k]*V[k].y;
	}

	V_new[t] = V[t];

	for(int s = 0; s < m; s++) {
		V_new[t].x += dt*c[order[s]]*F[t*m+s].x;	
		V_new[t].y += dt*c[order[s]]*F[t*m+s].y;	
	
	}

}